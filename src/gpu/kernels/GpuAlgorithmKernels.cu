#include "hip/hip_runtime.h"
#include "GpuAlgorithmKernels.cuh"
/*#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>*/

namespace cvt { 
namespace gpu {

// ==========================================================
//Interpolate the texture at point x, y 
// ==========================================================
__forceinline__ __device__ float texture_interpolateShort(const texture<short, hipTextureType2D, hipReadModeElementType> texref, float x, float y)
{
	int ix = (int)x; 
	int iy = (int)y;
	float dx = x - (float)ix;
	float dy = y - (float)iy;
	
	return(((float)tex2D(texref, ix, iy)*(1.f-dx)+(float)tex2D(texref, ix+1, iy)*(dx))*(1.f-dy)+((float)tex2D(texref, ix+1, iy+1)*(dx)+(float)tex2D(texref, ix, iy+1)*(1.f-dx))*(dy));
}

// ==========================================================
//	texture reference bind / un-bind
// ==========================================================

hipError_t bindTexture_sdsk_shortTileOne(hipArray * gpu_input_data)
{
	sdsk_shortTileOne.addressMode[0] = hipAddressModeClamp;
	sdsk_shortTileOne.addressMode[1] = hipAddressModeClamp;
	sdsk_shortTileOne.filterMode = hipFilterModePoint;
	sdsk_shortTileOne.normalized = false;
	hipBindTextureToArray(sdsk_shortTileOne, gpu_input_data);

	return hipGetLastError();
}

hipError_t unbindTexture_sdsk_shortTileOne()
{
	hipUnbindTexture(sdsk_shortTileOne);

	return hipGetLastError();
}

hipError_t bindTexture_sdsk_shortTileTwo(hipArray * gpu_input_data)
{
	sdsk_shortTileTwo.addressMode[0] = hipAddressModeClamp;
	sdsk_shortTileTwo.addressMode[1] = hipAddressModeClamp;
	sdsk_shortTileTwo.filterMode = hipFilterModePoint;
	sdsk_shortTileTwo.normalized = false;
	hipBindTextureToArray(sdsk_shortTileTwo, gpu_input_data);

	return hipGetLastError();
}

hipError_t unbindTexture_sdsk_shortTileTwo()
{
	hipUnbindTexture(sdsk_shortTileTwo);

	return hipGetLastError();
}


hipError_t bindTexture_sdsk_floatHueSaturation(hipArray * gpu_input_data)
{
	sdsk_floatHueSaturation.addressMode[0] = hipAddressModeClamp;
	sdsk_floatHueSaturation.addressMode[1] = hipAddressModeClamp;
	sdsk_floatHueSaturation.filterMode = hipFilterModePoint;
	sdsk_floatHueSaturation.normalized = false;
	hipBindTextureToArray(sdsk_floatHueSaturation, gpu_input_data);

	return hipGetLastError();
}

hipError_t bindTexture_sdsk_shortTwoDNormalized(hipArray * gpu_input_data)
{
	// ====================================================
	//	Bind the cuda array input to the texture
	// 		clamp indexing to border pixels of texture
	// ====================================================
	sdsk_shortTwoDNormalized.addressMode[0] = hipAddressModeClamp;
	sdsk_shortTwoDNormalized.addressMode[1] = hipAddressModeClamp;
	// 		Perform hardware linear interpolation
	sdsk_shortTwoDNormalized.filterMode = hipFilterModeLinear;
	// 		NOTE: normalized = true makes indexing into texture [0,1) instead of [0,N)
	sdsk_shortTwoDNormalized.normalized = false;

	// Bind Texture
	// The change Description from the
	hipBindTextureToArray(sdsk_floatHueSaturation, gpu_input_data);

	return hipGetLastError();
}

hipError_t unbindTexture_sdsk_shortTwoDNormalized()
{
	hipUnbindTexture(sdsk_shortTwoDNormalized);

	return hipGetLastError();
}

hipError_t bindTexture_sdsk_shortTwoD(hipArray * gpu_input_data)
{
	// ====================================================
	//	Bind the cuda array input to the texture
	// 		clamp indexing to border pixels of texture
	// ====================================================
	sdsk_shortTwoD.addressMode[0] = hipAddressModeClamp;
	sdsk_shortTwoD.addressMode[1] = hipAddressModeClamp;
	// 		Perform hardware linear interpolation
	sdsk_shortTwoD.filterMode = hipFilterModePoint;
	// 		NOTE: normalized = true makes indexing into texture [0,1) instead of [0,N)
	sdsk_shortTwoD.normalized = false;

	// Bind Texture
	// The change Description from the
	hipBindTextureToArray(sdsk_shortTwoD, gpu_input_data);

	return hipGetLastError();
}

hipError_t unbindTexture_sdsk_shortTwoD()
{
	hipUnbindTexture(sdsk_shortTwoD);

	return hipGetLastError();
}

hipError_t bindTexture_sdsk_ushortTwoD(hipArray * gpu_input_data)
{
	// ====================================================
	//	Bind the cuda array input to the texture
	// 		clamp indexing to border pixels of texture
	// ====================================================
	sdsk_ushortTwoD.addressMode[0] = hipAddressModeClamp;
	sdsk_ushortTwoD.addressMode[1] = hipAddressModeClamp;
	// 		Perform hardware linear interpolation
	sdsk_ushortTwoD.filterMode = hipFilterModePoint;
	// 		NOTE: normalized = true makes indexing into texture [0,1) instead of [0,N)
	sdsk_ushortTwoD.normalized = false;

	// Bind Texture
	// The change Description from the
	hipBindTextureToArray(sdsk_ushortTwoD, gpu_input_data);

	return hipGetLastError();
}

hipError_t unbindTexture_sdsk_ushortTwoD()
{
	hipUnbindTexture(sdsk_ushortTwoD);

	return hipGetLastError();
}


/*Explicit instantiations for data copy kernels */

template void launch_simpleDataCopy<int, int>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, int * in_data, 
						int * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<unsigned int, unsigned int>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, unsigned int * in_data, 
						unsigned int * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<unsigned char, unsigned char>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, unsigned char * in_data, 
						unsigned char * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<long, long>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, long * in_data, 
						long * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<unsigned short, unsigned short>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, unsigned short * in_data, 
						unsigned short * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<char, char>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, char * in_data, 
						char * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<float, float>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, float * in_data, 
						float * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<double, double>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, double * in_data, 
						double * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<short, short>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, short * in_data, 
						short * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<unsigned long, unsigned long>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, unsigned long * in_data, 
						unsigned long * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_simpleDataCopy<signed char, signed char>(dim3 dimGrid, dim3 dimBlock, unsigned int shmemSize, hipStream_t stream, signed char * in_data, 
						signed char * gpu_output_data, unsigned int outputWidth,  unsigned int outputHeight, unsigned int bandCount);

template void launch_window_histogram_statistics<short, float>(const dim3 dimGrid, const dim3 dimBlock, const unsigned int shmemSize,
		   const hipStream_t stream,  float * const outputData,
		   const unsigned int width,  const unsigned int height, int2 * const relativeOffsets,
		   const unsigned int numElements);


template void launch_dilate<short,short>(const dim3 dimGrid, const dim3 dimBlock, const unsigned int shmemSize, 
		   const hipStream_t stream,  short * const outputData, 
		   const unsigned int width,  const unsigned int height, int2 * const relativeOffsets, 
		   const unsigned int numElements);

template void launch_erode<short,short>(const dim3 dimGrid, const dim3 dimBlock, const unsigned int shmemSize, 
		   const hipStream_t stream,  short * const outputData, 
		   const unsigned int width,  const unsigned int height, int2 * const relativeOffsets, 
		   const unsigned int numElements);


}; //end gpu namespace
}; //end cvt namespace
